#include "hip/hip_runtime.h"
#include "bm3d.h"

/*
 * Read-only variables for all cuda kernels. These variables
 * will be stored in the "constant" memory on GPU for fast read.
 */
__constant__ GlobalConstants cu_const_params;

#include "block_matching.cu_inl"


float norm2(hipComplex & a) {
    return (a.x * a.x) + (a.y * a.y);
}

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////


__global__ void kernel() {
    printf("Here in kernel\n");
    printf("Image width: %d, height: %d\n", cu_const_params.image_width, cu_const_params.image_height);
}

__global__ void fill_precompute_data(hipfftComplex* precompute_patches) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int width = (cu_const_params.image_width - cu_const_params.patch_size + 1);
    int height = (cu_const_params.image_height - cu_const_params.patch_size + 1);
    if (i >= width || j >= height) {
        return;
    }
    // (i,j) is the top left corner of the patch
    for (int q=j;q<j+cu_const_params.patch_size;q++) {
        for (int p=i;p<i+cu_const_params.patch_size;p++) {
            // (p,q) is the image pixel
            int z = idx2(p-i,q-j,cu_const_params.patch_size);
            int index = idx3(z, i, j, cu_const_params.patch_size*cu_const_params.patch_size, width);
            precompute_patches[index].x = (float)(cu_const_params.image_data[idx2(p, q, cu_const_params.image_width)]);
            precompute_patches[index].y = 0.0f;
        }
    }
}


__global__ void real2complex(uchar* h_data, hipfftComplex *output) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = j*cu_const_params.image_width + i;

    if (i<cu_const_params.image_width && j<cu_const_params.image_height) {
        output[index].x = h_data[index];
        output[index].y = 0.0f;
    }
}

__global__ void complex2real(hipfftComplex *data, float* output, int total_size, int trans_size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= total_size) {
        return;
    }
    output[index] = data[index].x / (float)(trans_size);
}

/*
 *  normalize cufft inverse result by dividing number of elements per batch
 */
__global__ void normalize(hipfftComplex *data, int size) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = idx2(i, j, cu_const_params.image_width);
    data[index].x = data[index].x / (float)(size);
    data[index].y = data[index].y / (float)(size);
}

/*
 * taking d_rearrange_stacks and perform thresholding. Count number of non zeros
 * Also will normalize the 1D transform result.
 */
__global__ void hard_filter(hipfftComplex *d_rearrange_stacks, float *d_weight) {
    int group_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (group_id >= cu_const_params.total_ref_patches) {
        return;
    }
    int non_zero = 0;
    float threshold = cu_const_params.lambda_3d * cu_const_params.lambda_3d *
                      cu_const_params.sigma * cu_const_params.sigma *
                      blockIdx.x * blockIdx.y;
    int patch_size = cu_const_params.patch_size;
    int offset = group_id*cu_const_params.max_group_size * patch_size * patch_size;
    int norm_factor = cu_const_params.max_group_size;
    float x, y, val;
    for (int i=0; i<patch_size*patch_size*cu_const_params.max_group_size;i++) {
        x = d_rearrange_stacks[offset + i].x;
        y = d_rearrange_stacks[offset + i].y;
        x = x / norm_factor;
        y = y / norm_factor;
        val = x*x + y*y;
        if (val < threshold) {
            x = 0.0f;
            y = 0.0f;
        } else {
            ++non_zero;
        }
        d_rearrange_stacks[offset + i].x = x;
        d_rearrange_stacks[offset + i].y = y;
    }
    d_weight[group_id] = 1.0f / (float)non_zero;
}

/*
 *  Each thread maps to a group, d_transformed_stacks is organized as (w, h, patch in group)
 */
__global__ void fill_patch_major_from_source(Q* d_stacks, uint* d_num_patches_in_stack, uchar* input_data, hipfftComplex* d_transformed_stacks) {
    int group_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (group_id >= cu_const_params.total_ref_patches) {
        return;
    }
    int width = cu_const_params.image_width;
    int patch_size = cu_const_params.patch_size;

    // start patch num
    int start = group_id*cu_const_params.max_group_size;
    int offset = start * patch_size * patch_size;

    for (int z=0;z<d_num_patches_in_stack[group_id];z++) {
        // fill in the actual data
        uint patch_x = d_stacks[z+start].position.x;
        uint patch_y = d_stacks[z+start].position.y;
        for (int k=0;k<patch_size*patch_size;k++) {
            int index = idx2(patch_x + (k%patch_size), patch_y + (k/patch_size), width);
            int output_index = idx2(k, z, patch_size*patch_size);
            d_transformed_stacks[output_index+offset].x = (float)(input_data[index]);
        }
    }
}

/*
 *  Each thread maps to a group
 */
__global__ void fill_stack_major_data(hipfftComplex* d_transformed_stacks, hipfftComplex* d_rearrange_stacks) {
    int group_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (group_id >= cu_const_params.total_ref_patches) {
        return;
    }
    int patch_size = cu_const_params.patch_size;

    // start patch num
    int start = group_id*cu_const_params.max_group_size;
    int offset = start * patch_size * patch_size;

    for (int z=0;z<cu_const_params.max_group_size;z++) {
        for (int k=0;k<patch_size*patch_size;k++) {
            int w = k % patch_size;
            int h = k / patch_size;
            int output_index = idx3(z, w, h, cu_const_params.max_group_size, patch_size);
            int index = idx2(k, z, patch_size*patch_size);
            d_rearrange_stacks[output_index + offset].x = d_transformed_stacks[index + offset].x;
            d_rearrange_stacks[output_index + offset].y = d_transformed_stacks[index + offset].y;
        }
    }
}

__global__ void fill_patch_major_from_1D_layout(hipfftComplex* d_rearrange_stacks, hipfftComplex* d_transformed_stacks) {
    int group_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (group_id >= cu_const_params.total_ref_patches) {
        return;
    }
    int patch_size = cu_const_params.patch_size;

    // start patch num
    int start = group_id*cu_const_params.max_group_size;
    int offset = start * patch_size * patch_size;

    for (int i=0;i<patch_size*patch_size*cu_const_params.max_group_size;i++) {
        int h = (i / (cu_const_params.max_group_size * patch_size));
        int xz = i - h*cu_const_params.max_group_size * patch_size;
        int w = xz / cu_const_params.max_group_size;
        int z = xz % cu_const_params.max_group_size;
        int index = idx3(w, h, z, patch_size, patch_size);
        d_transformed_stacks[index+offset].x = d_rearrange_stacks[i+offset].x;
        d_transformed_stacks[index+offset].y = d_rearrange_stacks[i+offset].y;
    }
}

////////////////////////////////////////////////////////////////////////////////////////
// Class member functions
///////////////////////////////////////////////////////////////////////////////////////

/*
 * Initialize params struct
 */
Bm3d::Bm3d() {
    h_width = 0;
    h_height = 0;
    h_channels = 0;
    d_noisy_image = NULL;
    d_denoised_image = NULL;

    d_stacks = NULL;
    d_num_patches_in_stack = NULL;
    d_weight = NULL;
    d_wien_coef = NULL;
    d_kaiser_window = NULL;
}

Bm3d::~Bm3d() {

}

/*
 * Set first step params
 */
void Bm3d::set_fst_step_param() {

}

/*
 * Set second step params
 */
void Bm3d::set_2nd_step_param() {

}

/*
 * Set device params and allocate device memories
 */
void Bm3d::set_device_param(uchar* src_image) {
    int deviceCount = 0;
    total_patches = (h_width - h_fst_step_params.patch_size + 1) * (h_height - h_fst_step_params.patch_size + 1);
    total_ref_patches = ((h_width - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1) * ((h_height - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1);

    hipGetDeviceCount(&deviceCount);
    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);
    std::string name;
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // copy original image to cuda
    int size = h_width * h_height;
    hipMalloc(&d_noisy_image, sizeof(uchar) * h_channels * size);
    hipMemcpy(d_noisy_image, src_image, sizeof(uchar) * h_channels * size, hipMemcpyHostToDevice);

    hipMalloc(&precompute_patches, sizeof(hipfftComplex) * total_patches * h_fst_step_params.patch_size * h_fst_step_params.patch_size);
    hipMalloc(&d_stacks, sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size);
    hipMalloc(&d_num_patches_in_stack, sizeof(uint) * total_ref_patches);
    hipMalloc(&d_transformed_stacks, sizeof(hipfftComplex) * h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches);
    hipMalloc(&d_rearrange_stacks, sizeof(hipfftComplex) * h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches);
    hipMalloc(&d_weight, sizeof(float) * total_ref_patches);

    // Only use the generic params for now
    GlobalConstants params;
    params.image_width = h_width;
    params.image_height = h_height;
    params.image_data = d_noisy_image;
    params.image_channels = h_channels;

    params.patch_size = h_fst_step_params.patch_size;
    params.searching_window_size = h_fst_step_params.searching_window_size;
    params.stripe = h_fst_step_params.stripe;
    params.max_group_size = h_fst_step_params.max_group_size;
    params.distance_threshold_1 = h_fst_step_params.distance_threshold_1;
    params.distance_threshold_2 = h_fst_step_params.distance_threshold_2;
    params.sigma = h_fst_step_params.sigma;
    params.lambda_3d = h_fst_step_params.lambda_3d;
    params.beta = h_fst_step_params.beta;
    params.total_ref_patches = total_ref_patches;

    hipMemcpyToSymbol(HIP_SYMBOL(cu_const_params), &params, sizeof(GlobalConstants));
    int dim2D[2] = {h_fst_step_params.patch_size, h_fst_step_params.patch_size};
    // create cufft transform plan
    if(hipfftPlanMany(&plan, 2, dim2D,
                     NULL, 1, 0,
                     NULL, 1, 0,
                     HIPFFT_C2C, total_ref_patches*h_fst_step_params.max_group_size) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT Plan error: Plan failed");
        return;
    }
    int batch_size = total_ref_patches * h_fst_step_params.patch_size * h_fst_step_params.patch_size;
    if(hipfftPlan1d(&plan1D, h_fst_step_params.max_group_size, HIPFFT_C2C, batch_size) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT Plan error: Plan failed");
        return;
    }
}

/*
 * Initialize image stats and allocate memory
 */
void Bm3d::copy_image_to_device(uchar *src_image,
                                int width,
                                int height,
                                int channels) {
    // set width and height

}

void Bm3d::free_device_params() {
    if (d_noisy_image) {
        hipFree(d_noisy_image);
    }
}

/*
 * Take an image and run the algorithm to denoise.
 */
void Bm3d::denoise(uchar *src_image,
                   uchar *dst_image,
                   int width,
                   int height,
                   int channels,
                   int step,
                   int verbose = 1) {
    h_width = width;
    h_height = height;
    h_channels = channels;
    set_device_param(src_image);
    // precompute_2d_transform();
    denoise_fst_step();
    // fetch_data();
    // test_fill_precompute_data(src_image);
    // first step
    // test_cufft(src_image, dst_image);
    // DFT1D();
    // second step

    // copy image from device to host
    free_device_params();
}

/*
 * Perform the first step denoise
 */
void Bm3d::denoise_fst_step() {
    //Block matching, each thread maps to a ref patch
    do_block_matching();
    //gather patches, convert addresses to actual data

    arrange_block(d_noisy_image);

    //perform 2d dct transform
    // Stopwatch trans;
    // trans.start();
    if (hipfftExecC2C(plan, d_transformed_stacks, d_transformed_stacks, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }
    // trans.stop();
    // printf("Transform takes %.5f\n", trans.getSeconds());

    // transpose d_transformed_stacks to d_rearrange_stacks
    rearrange_to_1D_layout();
    // perform 1d transform
    if (hipfftExecC2C(plan1D, d_rearrange_stacks, d_rearrange_stacks, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }
    // hard thresholding and normalize
    hard_threshold();
    // inverse 1d transform
    if (hipfftExecC2C(plan1D, d_rearrange_stacks, d_rearrange_stacks, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }
    // transpose d_rearrange_stacks back to d_transformed_stacks
    rearrange_to_2D_layout();
    // inverse 2d transform
    // if (hipfftExecC2C(plan, d_transformed_stacks, d_transformed_stacks, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
    //     fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
    //     return;
    // }
    // aggregate to single image by writing into buffer
}

/*
 * Perform the second step denoise
 */
void Bm3d::denoise_2nd_step() {
    //Block matching estimate image, each thread maps to a ref patch

    //gather patches, convert addresses to actual data

    //gather noisy image patches, convert addresses to actual data

    // perform 2d dct transform on estimate

    // perform 1d transform on estimate

    // calculate Wiener coefficient for each group

    // apply wiener coefficient to each group of transformed noisy data

    // inverse 1d transform on noisy data

    // inverse 2d transform on noisy data

    // aggregate to single image by writing into buffer
}

void Bm3d::run_kernel() {
    kernel<<<1,1>>>();
}

/*
 * precompute the 2D transform on all the patches, the data is organized as follows:
 * for patch at (i,j) with patch size = 2, then in precompute_patches, the data is
 * stored as (i,j) (i+1,j) (i,j+1) (i+1,j+1), so the dimension is height*width*4
 * we first iterate z dim, then x dim then y dim.
 */
void Bm3d::precompute_2d_transform() {
    // prepare data
    Stopwatch fill_time;
    Stopwatch tran_time;
    int patch_size = h_fst_step_params.patch_size;
    int width = (h_width - patch_size + 1);
    int height = (h_height - patch_size + 1);
    int size = width*height*patch_size*patch_size;

    float* h_data = (float*)malloc(size*sizeof(float));
    dim3 dimBlock(16,16);
    dim3 dimGrid((width+15)/16, (height+15)/16);
    fill_time.start();
    fill_precompute_data<<<dimGrid, dimBlock>>>(precompute_patches);
    fill_time.stop();
    // 2D transformation
    tran_time.start();
    for(int i=0;i<width*height*patch_size*patch_size;i+=patch_size*patch_size*BATCH_2D) {
        if (hipfftExecC2C(plan, precompute_patches+i, precompute_patches+i, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
            return;
        }
    }
    tran_time.stop();
    printf("Data filling using %f\n", fill_time.getSeconds());
    printf("Exec using %f\n", tran_time.getSeconds());
}

void Bm3d::test_fill_precompute_data(uchar* src_image) {
    int patch_size = h_fst_step_params.patch_size;
    int width = (h_width - patch_size + 1);
    int height = (h_height - patch_size + 1);
    int size = width*height*patch_size*patch_size;
    float2* d_data;
    float2* h_data = (float2*)malloc(size*sizeof(float2));
    hipMalloc(&d_data, sizeof(float2) * size);

    dim3 dimBlock(16,16);
    dim3 dimGrid((width+15)/16, (height+15)/16);
    fill_precompute_data<<<dimGrid, dimBlock>>>((hipfftComplex*)d_data);
    hipMemcpy(h_data, d_data, size * sizeof(float2), hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed results copy\n");
        return;
    }
    inspect_patch(src_image, h_data, width, height, 0,0);
}

void Bm3d::inspect_patch(uchar* src_image, float2* h_data, int width, int height, int i, int j) {
    int p2 = h_fst_step_params.patch_size*h_fst_step_params.patch_size;
    h_data = h_data + j*width*p2 + i*p2;
    for (int q=j;q<j+h_fst_step_params.patch_size;q++) {
        for (int p=i;p<i+h_fst_step_params.patch_size;p++) {
            // (p,q) is the image pixel
            printf("Image Data: %zu, test data: %0.f\n", src_image[idx2(p,q,h_width)], (*h_data).x);
            h_data++;
        }
    }
}

void Bm3d::test_cufft(uchar* src_image, uchar* dst_image) {

}

void Bm3d::test_block_matching(uchar *input_image, int width, int height) {
    // generate a dummy image
    printf("testing block_matching\n");
    if (!input_image) {
        const int img_width = 40; // a 40 by 40 checkerboard of 8x8 patch
        const int patch_width = 8;
        uchar *dummy_image = (uchar *)malloc(img_width * img_width * sizeof(uchar));
        bool isWhite = false;
        for (int y = 0; y < img_width; y += patch_width) {
            for (int x = 0; x < img_width; x += patch_width) {
                // (x, y) is the top-left corner coordinate
                for (int j = 0; j < patch_width; ++j) {
                    for (int i = 0; i < patch_width; ++i) {
                        // (x + i, y + j) is the pixel coordinate
                        int idx = idx2(x+i, y+j, img_width);
                        input_image[idx] = isWhite ? 255 : 0;
                    }
                }
                isWhite = !isWhite;
            }
        }

        // set up the parameters and consts
        input_image = dummy_image;
    }
    h_width = width;
    h_height = height;
    h_channels = 1;
    set_device_param(input_image);

    printf("width, height: %d %d\n", width, height);

    // determine how many threads we need to spawn
    const int num_ref_patches_x = (h_width - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1;
    const int total_ref_patches = ((h_width - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1) * ((h_height - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1);
    printf("total_ref_patches %d\n", total_ref_patches);
    const int total_num_threads = total_ref_patches;
    const int threads_per_block = 256;
    const int num_blocks = (total_num_threads + threads_per_block - 1) / threads_per_block;
    printf("total_num_threads %d num_block %d\n", total_ref_patches, num_blocks);

    // hipError_t code = hipGetLastError();
    // if (code != hipSuccess) {
    //     fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(code));
    //     return;
    // }
    // call our block matching magic
    block_matching<<<num_blocks, threads_per_block>>>(d_stacks, d_num_patches_in_stack);
    Q *h_stacks = (Q *)malloc(sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size);
    hipMemcpy(h_stacks, d_stacks, sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size, hipMemcpyDeviceToHost);
    uint *h_num_patches_in_stack = (uint *)malloc(sizeof(uint) * total_ref_patches);
    hipMemcpy(h_num_patches_in_stack, d_num_patches_in_stack, sizeof(uint) * total_ref_patches, hipMemcpyDeviceToHost);

    // print the first stack
    const int which_stack = 13970;
    const int stack_x = which_stack % num_ref_patches_x;
    const int stack_y = which_stack / num_ref_patches_x;

    h_stacks = &h_stacks[which_stack * h_fst_step_params.max_group_size];



    printf("number of patches in stack %d: %d\n", which_stack, h_num_patches_in_stack[which_stack]);
    for (int i = 0; i < h_num_patches_in_stack[which_stack]; ++i) {
        const uint start_x = h_stacks[i].position.x;
        const uint start_y = h_stacks[i].position.y;
        printf("distance %d, x %d y %d\n", h_stacks[i].distance, start_x, start_y);
        for (int y = 0; y < h_fst_step_params.patch_size; ++y) {
            for (int x = 0; x < h_fst_step_params.patch_size; ++x) {
                const int idx = idx2( start_x + x, start_y + y, width);
                input_image[idx] = 255;
            }
        }
    }

    // set the original ref patch to 0
    for (int y = 0; y < h_fst_step_params.patch_size; ++y) {
        for (int x = 0; x < h_fst_step_params.patch_size; ++x) {
            const int idx = idx2(
                stack_x * h_fst_step_params.stripe + x,
                stack_y * h_fst_step_params.stripe + y,
                width);
            input_image[idx] = 0;
        }
    }

    // for (int y = 0; y < img_width; y += 1) {
    //     for (int x = 0; x < img_width; x += 1) {
    //         int idx = idx2(x, y, img_width);
    //         switch(input_image[idx]) {
    //             case 255:
    //                 printf("x");
    //                 break;
    //             case 127:
    //                 printf("o");
    //                 break;
    //             case 110:
    //                 printf("*");
    //                 break;
    //             default:
    //                 printf(" ");
    //         }
    //     }
    //     printf("\n");
    // }

    free_device_params();
}

/*
 *  arrange_block - according to the stacked patch indices, fetching data from the transformed
 *                  data array of 2D DCT. Input is an array of uint2, every N uint2
 *                  is a group. This kernel will put each group into an continuous array
 *                  of hipfftComplex num with x component to be the value, y component to be 0.f
 */
void Bm3d::arrange_block(uchar* input_data) {
    // input: Q* each struct is a patch with top left index
    // output: d_transformed_stacks, each patch got patch*patch size continuous chunk
    // each group will be assigned a thread
    Stopwatch arrange;
    arrange.start();
    int thread_per_block = 512;
    int num_blocks = (total_ref_patches + thread_per_block - 1) / thread_per_block;
    fill_patch_major_from_source<<<num_blocks, thread_per_block>>>(d_stacks, d_num_patches_in_stack, input_data, d_transformed_stacks);
    hipDeviceSynchronize();
    arrange.stop();
    printf("Arrange block takes %f\n", arrange.getSeconds());
}

void Bm3d::test_arrange_block(uchar *input_data) {
    int size = h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches;

    Q* test_q = (Q*)malloc(sizeof(Q)*total_ref_patches * h_fst_step_params.max_group_size);
    for (int i=0;i<2*h_fst_step_params.max_group_size; i++) {
        test_q[i].position.x = i;
        test_q[i].position.y = 0;
    }
    float* h_data = (float*)malloc(sizeof(float) * size);
    float* d_data;
    hipMalloc(&d_data, sizeof(float) * size);
    hipMemcpy(d_stacks, test_q, sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size, hipMemcpyHostToDevice);
    uint* h_num_patches = (uint*)calloc(total_ref_patches, sizeof(uint));
    h_num_patches[0] = h_fst_step_params.max_group_size;
    h_num_patches[1] = h_fst_step_params.max_group_size - 2;
    hipMemcpy(d_num_patches_in_stack, h_num_patches, sizeof(uint)*total_ref_patches, hipMemcpyHostToDevice);
    arrange_block(d_noisy_image);

    if (hipfftExecC2C(plan, d_transformed_stacks, d_transformed_stacks, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }

    if (hipfftExecC2C(plan, d_transformed_stacks, d_transformed_stacks, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }
    int threads_per_block = 512;
    int num_blocks = (size + threads_per_block - 1) / threads_per_block;
    complex2real<<<num_blocks, threads_per_block>>>(d_transformed_stacks, d_data, size, h_fst_step_params.patch_size*h_fst_step_params.patch_size);

    hipMemcpy(h_data, d_data, size * sizeof(float), hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed results copy\n");
        return;
    }
    for (int i=0;i<2*h_fst_step_params.patch_size*h_fst_step_params.patch_size*h_fst_step_params.max_group_size;i++) {
        int x = i/(h_fst_step_params.patch_size*h_fst_step_params.patch_size);
        int y = 0;
        if (i % (h_fst_step_params.patch_size*h_fst_step_params.patch_size) == 0) {
            printf("Patch (%d, %d)\n", x, 0);
        }
        int z = i - x*(h_fst_step_params.patch_size*h_fst_step_params.patch_size);
        int index = idx2(x+(z%h_fst_step_params.patch_size), y+(z/h_fst_step_params.patch_size), h_width);
        printf("Transform: %.3f vs Original: %zu\n",
            h_data[i],
            input_data[index]
            );
    }
}

/*
 * fetch_data - according to the stacked patch indices, fetching data from the transformed
 *              data array of 2D DCT. Input is an array of uint2, every N uint2
 *              is a group. For each group of dim (width, height, num_patches), we will go
 *              through num_pathches first, then width then height.
 */
void Bm3d::rearrange_to_1D_layout() {
    Stopwatch fetch;
    fetch.start();
    int thread_per_block = 512;
    int num_blocks = (total_ref_patches + thread_per_block - 1) / thread_per_block;
    fill_stack_major_data<<<num_blocks, thread_per_block>>>(d_transformed_stacks, d_rearrange_stacks);
    hipDeviceSynchronize();
    fetch.stop();
    printf("rearrange_to_1D_layout takes %.5f\n", fetch.getSeconds());
}

void Bm3d::rearrange_to_2D_layout() {
    Stopwatch fetch;
    fetch.start();
    int thread_per_block = 512;
    int num_blocks = (total_ref_patches + thread_per_block - 1) / thread_per_block;
    fill_patch_major_from_1D_layout<<<num_blocks, thread_per_block>>>(d_rearrange_stacks, d_transformed_stacks);
    hipDeviceSynchronize();
    fetch.stop();
    printf("rearrange_to_2D_layout takes %.5f\n", fetch.getSeconds());
}

/*
 * do_block_matching - launch kernel to run block matching
 */
void Bm3d::do_block_matching() {
    // determine how many threads we need to spawn
    Stopwatch bm_time;
    bm_time.start();
    printf("total_ref_patches %d\n", total_ref_patches);
    const int total_num_threads = total_ref_patches;
    const int threads_per_block = 256;
    const int num_blocks = (total_num_threads + threads_per_block - 1) / threads_per_block;
    printf("total_num_threads %d num_block %d\n", total_ref_patches, num_blocks);
    block_matching<<<num_blocks, threads_per_block>>>(d_stacks, d_num_patches_in_stack);
    hipDeviceSynchronize();
    bm_time.stop();
    printf("Block Matching: %f\n", bm_time.getSeconds());
}

void Bm3d::hard_threshold() {
    Stopwatch hard_threshold;
    hard_threshold.start();
    int thread_per_block = 512;
    int num_blocks = (total_ref_patches + thread_per_block - 1) / thread_per_block;
    hard_filter<<<num_blocks, thread_per_block>>>(d_rearrange_stacks, d_weight);
    hipDeviceSynchronize();
    hard_threshold.stop();
    printf("Hard threshold takes %.5f\n", hard_threshold.getSeconds());
}
