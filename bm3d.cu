#include "hip/hip_runtime.h"
#include "bm3d.h"

/*
 * Read-only variables for all cuda kernels. These variables
 * will be stored in the "constant" memory on GPU for fast read.
 */
__constant__ GlobalConstants cu_const_params;

float abspow2(hipComplex & a)
{
    return (a.x * a.x) + (a.y * a.y);
}

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////


__global__ void kernel() {
    printf("Here in kernel\n");
    printf("Image width: %d, height: %d\n", cu_const_params.image_width, cu_const_params.image_height);
}

__global__ void real2complex(uchar* h_data, hipfftComplex *output) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = j*cu_const_params.image_width + i;

    if (i<cu_const_params.image_width && j<cu_const_params.image_height) {
        output[index].x = h_data[index];
        output[index].y = 0.0f;
    }
}

__global__ void complex2real(hipfftComplex *data, uchar* output) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = j*cu_const_params.image_width + i;
    int size = cu_const_params.image_width*cu_const_params.image_height;

    if (i<cu_const_params.image_width && j<cu_const_params.image_height) {
        output[index] = data[index].x / (float)(size);
    }
}

////////////////////////////////////////////////////////////////////////////////////////
// Class member functions
///////////////////////////////////////////////////////////////////////////////////////

/*
 * Initialize params struct
 */
Bm3d::Bm3d() {
    h_width = 0;
    h_height = 0;
    h_channels = 0;
    d_noisy_image = NULL;
    d_denoised_image = NULL;

    d_stacks = NULL;
    d_num_patches_in_stack = NULL;
    d_weight = NULL;
    d_wien_coef = NULL;
    d_kaiser_window = NULL;
}

Bm3d::~Bm3d() {

}

/*
 * Set first step params
 */
void Bm3d::set_fst_step_param() {

}

/*
 * Set second step params
 */
void Bm3d::set_2nd_step_param() {

}

/*
 * Set device params and allocate device memories
 */
void Bm3d::set_device_param(uchar* src_image) {
    int deviceCount = 0;
    hipError_t err;
    err = hipGetDeviceCount(&deviceCount);
    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);
    std::string name;
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // copy original image to cuda
    int size = h_width * h_height;
    hipMalloc(&d_noisy_image, sizeof(uchar) * h_channels * size);
    hipMemcpy(d_noisy_image, src_image, sizeof(uchar) * h_channels * size, hipMemcpyHostToDevice);

    // Only use the generic params for now
    GlobalConstants params;
    params.image_width = h_width;
    params.image_height = h_height;
    params.image_data = d_noisy_image;
    params.image_channels = h_channels;

    params.patch_size = h_fst_step_params.patch_size;
    params.searching_window_size = h_fst_step_params.searching_window_size;
    params.stripe = h_fst_step_params.stripe;
    params.max_group_size = h_fst_step_params.max_group_size;
    params.distance_threshold_1 = h_fst_step_params.distance_threshold_1;
    params.distance_threshold_2 = h_fst_step_params.distance_threshold_2;
    params.sigma = h_fst_step_params.sigma;
    params.lambda_3d = h_fst_step_params.lambda_3d;
    params.beta = h_fst_step_params.beta;
    printf("params: %d, %d\n", params.image_width, params.image_height);

    err = hipMemcpyToSymbol(HIP_SYMBOL(cu_const_params), &params, sizeof(GlobalConstants));

    printf("%s\n", hipGetErrorString(err));
}

/*
 * Initialize image stats and allocate memory
 */
void Bm3d::copy_image_to_device(uchar *src_image,
                                int width,
                                int height,
                                int channels) {
    // set width and height

}

void Bm3d::free_device_params() {
    if (d_noisy_image) {
        hipFree(d_noisy_image);
    }
}

/*
 * Take an image and run the algorithm to denoise.
 */
void Bm3d::denoise(uchar *src_image,
                   uchar *dst_image,
                   int width,
                   int height,
                   int channels,
                   int step,
                   int verbose = 1) {
    h_width = width;
    h_height = height;
    h_channels = channels;
    set_device_param(src_image);
    // first step
    test_cufft(src_image, dst_image);
    // second step

    // copy image from device to host
    free_device_params();
}

/*
 * Perform the first step denoise
 */
void Bm3d::denoise_fst_step() {

}

/*
 * Perform the second step denoise
 */
void Bm3d::denoise_2nd_step() {

}

void Bm3d::run_kernel() {
    kernel<<<1,1>>>();
}

void Bm3d::test_cufft(uchar* src_image, uchar* dst_image) {
    Stopwatch init_time;
    Stopwatch exec_time;
    init_time.start();
    int size = h_width * h_height;

    hipfftHandle plan;
    uchar *h_data;
    uchar *d_data;
    hipMalloc(&d_data, sizeof(uchar) * size);

    hipMalloc(&h_data, sizeof(uchar) * size);
    hipMemcpy(h_data, src_image, sizeof(uchar) * size, hipMemcpyHostToDevice);

    hipfftComplex *data;
    hipMalloc(&data, sizeof(hipfftComplex) * size);
    int n[2] = {16,16};

    if(hipfftPlanMany(&plan, 2, n
                     NULL, 1, 0,
                     NULL, 1, 0,
                     HIPFFT_C2C, size/256) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT Plan error: Plan failed");
        return;
    }
    init_time.stop();
    exec_time.start();
    // get input in shape
    dim3 dimBlock(16,16);
    dim3 dimGrid(h_width/16, h_height/16);
    real2complex<<<dimGrid, dimBlock>>>(h_data, data);

    if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }

    if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }
    complex2real<<<dimGrid, dimBlock>>>(data, d_data);
    hipMemcpy(dst_image, d_data, size * sizeof(uchar), hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed results copy\n");
        return;
    }
    exec_time.stop();
    printf("Init: %f\n", init_time.getSeconds());
    printf("Exec: %f\n", exec_time.getSeconds());
    for (int i=0;i<size;i++) {
        printf("%d: (%zu, %zu)\n", i, src_image[i], dst_image[i]);
    }
}
