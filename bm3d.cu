#include "hip/hip_runtime.h"
#include "bm3d.h"

/*
 * Read-only variables for all cuda kernels. These variables
 * will be stored in the "constant" memory on GPU for fast read.
 */
__constant__ GlobalConstants cu_const_params;

//#include "block_matching.cu_inl"

float abspow2(hipComplex & a)
{
    return (a.x * a.x) + (a.y * a.y);
}

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////
__device__ float norm2(hipComplex & a) {
    return (a.x * a.x) + (a.y * a.y);
}

__global__ void kernel() {
    printf("Here in kernel\n");
    printf("Image width: %d, height: %d\n", cu_const_params.image_width, cu_const_params.image_height);
}

__global__ void fill_precompute_data(hipfftComplex* precompute_patches) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int width = (cu_const_params.image_width - cu_const_params.patch_size + 1);
    int height = (cu_const_params.image_height - cu_const_params.patch_size + 1);
    if (i >= width || j >= height) {
        return;
    }
    // (i,j) is the top left corner of the patch
    for (int q=j;q<j+cu_const_params.patch_size;q++) {
        for (int p=i;p<i+cu_const_params.patch_size;p++) {
            // (p,q) is the image pixel
            int z = idx2(p-i,q-j,cu_const_params.patch_size);
            int index = idx3(z, i, j, cu_const_params.patch_size*cu_const_params.patch_size, width);
            precompute_patches[index].x = (float)(cu_const_params.image_data[idx2(p, q, cu_const_params.image_width)]);
            precompute_patches[index].y = 0.0f;
        }
    }
}


__global__ void real2complex(uchar* h_data, hipfftComplex *output) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = j*cu_const_params.image_width + i;

    if (i<cu_const_params.image_width && j<cu_const_params.image_height) {
        output[index].x = h_data[index];
        output[index].y = 0.0f;
    }
}

__global__ void complex2real(hipfftComplex *data, uchar* output, int size) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = j*cu_const_params.image_width + i;

    if (i<cu_const_params.image_width && j<cu_const_params.image_height) {
        output[index] = data[index].x / (float)(size);

    }
}

/*
 *  normalize cufft inverse result by dividing number of elements per batch
 */
__global__ void normalize(hipfftComplex *data, int size) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = idx2(i, j, cu_const_params.image_width);
    data[index].x = data[index].x / (float)(size);
    data[index].y = data[index].y / (float)(size);
}

__global__ void hard_filter(hipfftComplex *data) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = idx2(i, j, cu_const_params.image_width);

    float threshold = cu_const_params.lambda_3d * cu_const_params.lambda_3d *
                      cu_const_params.sigma * cu_const_params.sigma *
                      blockIdx.x * blockIdx.y*10000;
    float val = norm2(data[index]);
    if (val < threshold) {
        data[index].x = 0.0f;
        data[index].y = 0.0f;
        // printf("index: %d with norm %f\n", index, val);
    }
}

/*
 *  Each thread maps to a group
 */
__global__ void fill_data(Q* d_stacks, uint* d_num_patches_in_stack, hipfftComplex* precompute_patches, hipfftComplex* d_transformed_stacks) {
    int group_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (group_id >= cu_const_params.total_ref_patches) {
        return;
    }
    int width = (cu_const_params.image_width - cu_const_params.patch_size + 1);
    int patch_size = cu_const_params.patch_size;

    // start patch num
    int start = group_id*cu_const_params.max_group_size;
    d_transformed_stacks += start * patch_size * patch_size;

    for (int i=start;i<start+cu_const_params.max_group_size;i++) {
        if (i - start < d_num_patches_in_stack[group_id]) {
            // fill in the actual data
            uint patch_x = d_stacks[i].position.x;
            uint patch_y = d_stacks[i].position.y;
            for (int z=0;z<patch_size*patch_size;z++) {
                int index = idx3(z, patch_x, patch_y, patch_size*patch_size, width);
                d_transformed_stacks->x = precompute_patches[index].x;
                d_transformed_stacks->y = precompute_patches[index].y;
                d_transformed_stacks++;
            }
        } else {
            // fill 0s
            for (int z=0;z<patch_size*patch_size;z++) {
                d_transformed_stacks->x = 0.0f;
                d_transformed_stacks->y = 0.0f;
                d_transformed_stacks++;
            }
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////
// Class member functions
///////////////////////////////////////////////////////////////////////////////////////

/*
 * Initialize params struct
 */
Bm3d::Bm3d() {
    h_width = 0;
    h_height = 0;
    h_channels = 0;
    d_noisy_image = NULL;
    d_denoised_image = NULL;

    d_stacks = NULL;
    d_num_patches_in_stack = NULL;
    d_weight = NULL;
    d_wien_coef = NULL;
    d_kaiser_window = NULL;
}

Bm3d::~Bm3d() {

}

/*
 * Set first step params
 */
void Bm3d::set_fst_step_param() {

}

/*
 * Set second step params
 */
void Bm3d::set_2nd_step_param() {

}

/*
 * Set device params and allocate device memories
 */
void Bm3d::set_device_param(uchar* src_image) {
    int deviceCount = 0;
    total_patches = (h_width - h_fst_step_params.patch_size + 1) * (h_height - h_fst_step_params.patch_size + 1);
    total_ref_patches = ((h_width - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1) * ((h_height - h_fst_step_params.patch_size) / h_fst_step_params.stripe + 1);

    hipGetDeviceCount(&deviceCount);
    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);
    std::string name;
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // copy original image to cuda
    int size = h_width * h_height;
    hipMalloc(&d_noisy_image, sizeof(uchar) * h_channels * size);
    hipMemcpy(d_noisy_image, src_image, sizeof(uchar) * h_channels * size, hipMemcpyHostToDevice);

    hipMalloc(&precompute_patches, sizeof(hipfftComplex) * total_patches * h_fst_step_params.patch_size * h_fst_step_params.patch_size);
    hipMalloc(&d_stacks, sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size);
    hipMalloc(&d_num_patches_in_stack, sizeof(uint) * total_ref_patches);
    hipMalloc(&d_transformed_stacks, sizeof(hipfftComplex) * h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches);


    // Only use the generic params for now
    GlobalConstants params;
    params.image_width = h_width;
    params.image_height = h_height;
    params.image_data = d_noisy_image;
    params.image_channels = h_channels;

    params.patch_size = h_fst_step_params.patch_size;
    params.searching_window_size = h_fst_step_params.searching_window_size;
    params.stripe = h_fst_step_params.stripe;
    params.max_group_size = h_fst_step_params.max_group_size;
    params.distance_threshold_1 = h_fst_step_params.distance_threshold_1;
    params.distance_threshold_2 = h_fst_step_params.distance_threshold_2;
    params.sigma = h_fst_step_params.sigma;
    params.lambda_3d = h_fst_step_params.lambda_3d;
    params.beta = h_fst_step_params.beta;
    params.total_ref_patches = total_ref_patches;

    hipMemcpyToSymbol(HIP_SYMBOL(cu_const_params), &params, sizeof(GlobalConstants));
    int dim2D[2] = {h_fst_step_params.patch_size, h_fst_step_params.patch_size};
    // create cufft transform plan
    if(hipfftPlanMany(&plan, 2, dim2D,
                     NULL, 1, 0,
                     NULL, 1, 0,
                     HIPFFT_C2C, BATCH_2D) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT Plan error: Plan failed");
        return;
    }
    int dim1D[1] = {h_fst_step_params.max_group_size};
    int inembed[1] = {0};
    int onembed[1] = {0};
    if(hipfftPlanMany(&plan1D, 1, dim1D,
                     inembed,
                     h_fst_step_params.patch_size* h_fst_step_params.patch_size, // stride
                     1, // batch distance
                     onembed,
                     h_fst_step_params.patch_size* h_fst_step_params.patch_size, // stride
                     1,
                     HIPFFT_C2C,
                     h_fst_step_params.patch_size* h_fst_step_params.patch_size // batch size
                     ) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT Plan error: Plan failed");
        return;
    }

}

/*
 * Initialize image stats and allocate memory
 */
void Bm3d::copy_image_to_device(uchar *src_image,
                                int width,
                                int height,
                                int channels) {
    // set width and height

}

void Bm3d::free_device_params() {
    if (d_noisy_image) {
        hipFree(d_noisy_image);
    }
}

/*
 * Take an image and run the algorithm to denoise.
 */
void Bm3d::denoise(uchar *src_image,
                   uchar *dst_image,
                   int width,
                   int height,
                   int channels,
                   int step,
                   int verbose = 1) {
    h_width = width;
    h_height = height;
    h_channels = channels;
    set_device_param(src_image);
    precompute_2d_transform();
    // test_fill_precompute_data(src_image);
    // first step
    // test_cufft(src_image, dst_image);
    DFT1D();
    // second step

    // copy image from device to host
    free_device_params();
}

/*
 * Perform the first step denoise
 */
void Bm3d::denoise_fst_step() {
    //Block matching, each thread maps to a ref patch

    //gather patches, convert addresses to actual data

    //perform 2d dct transform

    // perform 1d transform

    // hard thresholding

    // inverse 1d transform

    // inverse 2d transform

    // aggregate to single image by writing into buffer
}

/*
 * Perform the second step denoise
 */
void Bm3d::denoise_2nd_step() {
    //Block matching estimate image, each thread maps to a ref patch

    //gather patches, convert addresses to actual data

    //gather noisy image patches, convert addresses to actual data

    // perform 2d dct transform on estimate

    // perform 1d transform on estimate

    // calculate Wiener coefficient for each group

    // apply wiener coefficient to each group of transformed noisy data

    // inverse 1d transform on noisy data

    // inverse 2d transform on noisy data

    // aggregate to single image by writing into buffer
}

void Bm3d::run_kernel() {
    kernel<<<1,1>>>();
}

/*
 * precompute the 2D transform on all the patches, the data is organized as follows:
 * for patch at (i,j) with patch size = 2, then in precompute_patches, the data is
 * stored as (i,j) (i+1,j) (i,j+1) (i+1,j+1), so the dimension is height*width*4
 * we first iterate z dim, then x dim then y dim.
 */
void Bm3d::precompute_2d_transform() {
    // prepare data
    Stopwatch fill_time;
    Stopwatch tran_time;
    int patch_size = h_fst_step_params.patch_size;
    int width = (h_width - patch_size + 1);
    int height = (h_height - patch_size + 1);
    int size = width*height*patch_size*patch_size;

    float* h_data = (float*)malloc(size*sizeof(float));
    dim3 dimBlock(16,16);
    dim3 dimGrid((width+15)/16, (height+15)/16);
    fill_time.start();
    fill_precompute_data<<<dimGrid, dimBlock>>>(precompute_patches);
    fill_time.stop();
    // 2D transformation
    tran_time.start();
    for(int i=0;i<width*height*patch_size*patch_size;i+=patch_size*patch_size*BATCH_2D) {
        if (hipfftExecC2C(plan, precompute_patches+i, precompute_patches+i, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
            return;
        }
    }
    tran_time.stop();
    printf("Data filling using %f\n", fill_time.getSeconds());
    printf("Exec using %f\n", tran_time.getSeconds());
}

void Bm3d::test_fill_precompute_data(uchar* src_image) {
    int patch_size = h_fst_step_params.patch_size;
    int width = (h_width - patch_size + 1);
    int height = (h_height - patch_size + 1);
    int size = width*height*patch_size*patch_size;
    float2* d_data;
    float2* h_data = (float2*)malloc(size*sizeof(float2));
    hipMalloc(&d_data, sizeof(float2) * size);

    dim3 dimBlock(16,16);
    dim3 dimGrid((width+15)/16, (height+15)/16);
    fill_precompute_data<<<dimGrid, dimBlock>>>((hipfftComplex*)d_data);
    hipMemcpy(h_data, d_data, size * sizeof(float2), hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed results copy\n");
        return;
    }
    inspect_patch(src_image, h_data, width, height, 0,0);
}

void Bm3d::inspect_patch(uchar* src_image, float2* h_data, int width, int height, int i, int j) {
    int p2 = h_fst_step_params.patch_size*h_fst_step_params.patch_size;
    h_data = h_data + j*width*p2 + i*p2;
    for (int q=j;q<j+h_fst_step_params.patch_size;q++) {
        for (int p=i;p<i+h_fst_step_params.patch_size;p++) {
            // (p,q) is the image pixel
            printf("Image Data: %zu, test data: %0.f\n", src_image[idx2(p,q,h_width)], (*h_data).x);
            h_data++;
        }
    }
}

void Bm3d::test_cufft(uchar* src_image, uchar* dst_image) {
    Stopwatch init_time;
    Stopwatch exec_time;
    init_time.start();
    int size = h_width * h_height;
    int patch_size = h_fst_step_params.patch_size;
    int group_size = h_fst_step_params.max_group_size;;

    // hipfftHandle plan_tmp;
    // hipfftHandle plan1D_tmp;
    uchar *h_data;
    uchar *d_data;
    hipMalloc(&d_data, sizeof(uchar) * size);

    hipMalloc(&h_data, sizeof(uchar) * size);
    hipMemcpy(h_data, src_image, sizeof(uchar) * size, hipMemcpyHostToDevice);

    hipfftComplex *data;
    hipMalloc(&data, sizeof(hipfftComplex) * size);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: initialize error\n");
        return;
    }
    init_time.stop();
    exec_time.start();
    // get input in shape
    dim3 dimBlock(16,16);
    dim3 dimGrid(h_width/16, h_height/16);
    real2complex<<<dimGrid, dimBlock>>>(h_data, data);

    // batch size 2D transform. cufft batch size should be determined at plan time
    for (int i=0;i<size;i+=patch_size*patch_size*BATCH_2D) {
        if (hipfftExecC2C(plan, data+i, data+i, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
            return;
        }
    }

    for (int i=0;i<size;i+=patch_size*patch_size*group_size*BATCH_1D) {
        if (hipfftExecC2C(plan1D, data+i, data+i, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
            return;
        }
    }

    //hard filter
    // hard_filter<<<dimGrid, dimBlock>>>(data);

    for (int i=0;i<size;i+=patch_size*patch_size*group_size*BATCH_1D) {
        if (hipfftExecC2C(plan1D, data+i, data+i, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
            return;
        }
    }

    // normalize cufft 1d transformation
    normalize<<<dimGrid, dimBlock>>>(data, patch_size*patch_size*group_size);
    for (int i=0;i<size;i+=patch_size*patch_size*BATCH_2D) {
        if (hipfftExecC2C(plan, data+i, data+i, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
            return;
        }
    }

    complex2real<<<dimGrid, dimBlock>>>(data, d_data, patch_size*patch_size);

    hipMemcpy(dst_image, d_data, size * sizeof(uchar), hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed results copy\n");
        return;
    }
    exec_time.stop();
    printf("Init: %f\n", init_time.getSeconds());
    printf("Exec: %f\n", exec_time.getSeconds());
    for (int i=0;i<size;i++) {
        printf("%d: (%zu, %zu)\n", i, src_image[i], dst_image[i]);
    }
}

/*
 *  arrange_block - according to the stacked patch indices, fetching data from the transformed
 *                  data array of 2D DCT. Input is an array of uint2, every N uint2
 *                  is a group. This kernel will put each group into an continuous array
 *                  of hipfftComplex num with x component to be the value, y component to be 0.f
 */
void Bm3d::arrange_block() {
    // input: Q* each struct is a patch with top left index
    // output: d_transformed_stacks, each patch got patch*patch size continuous chunk
    // each group will be assigned a thread
    int thread_per_block = 256;
    int num_blocks = (total_ref_patches + thread_per_block - 1) / thread_per_block;
    fill_data<<<num_blocks, thread_per_block>>>(d_stacks, d_num_patches_in_stack, precompute_patches, d_transformed_stacks);
}

void Bm3d::test_arrange_block() {
    int size = h_fst_step_params.patch_size * h_fst_step_params.patch_size * h_fst_step_params.max_group_size * total_ref_patches;

    Q* test_q = (Q*)malloc(sizeof(Q)*total_ref_patches * h_fst_step_params.max_group_size);
    for (int i=0;i<2*h_fst_step_params.max_group_size; i++) {
        test_q[i].position.x = i;
        test_q[i].position.y = 0;
    }
    hipfftComplex* h_transformed_stacks = (hipfftComplex*)malloc(sizeof(hipfftComplex) * size);

    hipMemcpy(d_stacks, test_q, sizeof(Q) * total_ref_patches * h_fst_step_params.max_group_size, hipMemcpyHostToDevice);
    uint* h_num_patches = (uint*)calloc(total_ref_patches, sizeof(uint));
    h_num_patches[0] = h_fst_step_params.max_group_size;
    h_num_patches[1] = h_fst_step_params.max_group_size - 2;
    hipMemcpy(d_num_patches_in_stack, h_num_patches, sizeof(uint)*total_ref_patches, hipMemcpyHostToDevice);
    arrange_block();
    hipMemcpy(h_transformed_stacks, d_transformed_stacks, sizeof(hipfftComplex) * size, hipMemcpyDeviceToHost);
    float2* h_data = (float2*)malloc(size*sizeof(float2));
    hipMemcpy(h_data, (float2*)precompute_patches, size * sizeof(float2), hipMemcpyDeviceToHost);
    for (int i=0;i<2*h_fst_step_params.patch_size*h_fst_step_params.patch_size*h_fst_step_params.max_group_size;i++) {
        int x = i/(h_fst_step_params.patch_size*h_fst_step_params.patch_size);
        int y = 0;
        if (i % (h_fst_step_params.patch_size*h_fst_step_params.patch_size) == 0) {
            printf("Patch (%d, %d)\n", x, 0);
        }
        int z = i - x*(h_fst_step_params.patch_size*h_fst_step_params.patch_size);
        int index = idx3(z, x, y, h_fst_step_params.patch_size*h_fst_step_params.patch_size, h_width);
        printf("Transform: (%.3f, %.3f) vs Precompute: (%.3f, %.3f)\n",
            h_transformed_stacks[i].x,
            h_transformed_stacks[i].y,
            h_data[index].x,
            h_data[index].y);
    }
}

/*
 * DFT1D - Perform the 1D DFT transform on the 3D stacks. Since the data is organized
 *         as iterate through each patch in every group. We need to perform 1D DFT
 *         on the same pixel of every patch in the same group. We will use the stride.
 */
void Bm3d::DFT1D() {
    Stopwatch trans;
    trans.start();
    int step_size = h_fst_step_params.max_group_size * h_fst_step_params.patch_size * h_fst_step_params.patch_size;
    int total_size = total_ref_patches * step_size;
    for (int i=0; i<total_size; i+=step_size) {
        if (hipfftExecC2C(plan1D, d_transformed_stacks+i, d_transformed_stacks+i, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
            fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
            return;
        }
    }
    trans.stop();
    printf("1D transform needs %.5f\n", trans.getSeconds());
}

/*
 * do_block_matching - launch kernel to run block matching
 */
void Bm3d::do_block_matching(
    Q* g_stacks,                //OUT: Size [num_ref * max_num_patches_in_stack]
    uint* g_num_patches_in_stack   //OUT: For each reference patch contains number of similar patches. Size [num_ref]
    ) {
}
