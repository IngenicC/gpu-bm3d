#include "hip/hip_runtime.h"
#include "bm3d.h"

/*
 * Read-only variables for all cuda kernels. These variables
 * will be stored in the "constant" memory on GPU for fast read.
 */
__constant__ GlobalConstants cu_const_params;

float abspow2(hipComplex & a)
{
    return (a.x * a.x) + (a.y * a.y);
}

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////


__global__ void kernel() {
    printf("Here in kernel\n");
    printf("Image width: %d, height: %d\n", cu_const_params.image_width, cu_const_params.image_height);
}

__global__ void real2complex(uchar* h_data, hipfftComplex *output) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = j*cu_const_params.image_width + i;

    if (i<cu_const_params.image_width && j<cu_const_params.image_height) {
        output[index].x = h_data[index];
        output[index].y = 0.0f;
    }
}

__global__ void complex2real(hipfftComplex *data, uchar* output, int size) {
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int index = j*cu_const_params.image_width + i;

    if (i<cu_const_params.image_width && j<cu_const_params.image_height) {
        output[index] = data[index].x / (float)(size);
    }
}

__global__ void fill_data(uint2 *d_stacks, hipfftComplex *data_stack, int size, int patch_size, int group_size) {
    for (int i=0;i<group_size;i++) {
        int b_idx = blockIdx.x * group_size + i;
        int ref_x = d_stacks[b_idx].x;
        int ref_y = d_stacks[b_idx].y;

        int start_idx = b_idx * patch_size * patch_size;
        data_stack += start_idx;
        data_stack[idx2(threadIdx.x, threadIdx.y, patch_size)].x = (float)(cu_const_params.image_data[idx2(ref_x+threadIdx.x, ref_y+threadIdx.y, cu_const_params.image_width)]);
        data_stack[idx2(threadIdx.x, threadIdx.y, patch_size)].y = 0.0f;
        printf("idx: %d, %f\n", idx2(threadIdx.x, threadIdx.y, patch_size) + start_idx, data_stack[idx2(threadIdx.x, threadIdx.y, patch_size)].x);
    }

}

////////////////////////////////////////////////////////////////////////////////////////
// Class member functions
///////////////////////////////////////////////////////////////////////////////////////

/*
 * Initialize params struct
 */
Bm3d::Bm3d() {
    h_width = 0;
    h_height = 0;
    h_channels = 0;
    d_noisy_image = NULL;
    d_denoised_image = NULL;

    d_stacks = NULL;
    d_num_patches_in_stack = NULL;
    d_weight = NULL;
    d_wien_coef = NULL;
    d_kaiser_window = NULL;
}

Bm3d::~Bm3d() {

}

/*
 * Set first step params
 */
void Bm3d::set_fst_step_param() {

}

/*
 * Set second step params
 */
void Bm3d::set_2nd_step_param() {

}

/*
 * Set device params and allocate device memories
 */
void Bm3d::set_device_param(uchar* src_image) {
    int deviceCount = 0;
    hipError_t err;
    err = hipGetDeviceCount(&deviceCount);
    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);
    std::string name;
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // copy original image to cuda
    int size = h_width * h_height;
    hipMalloc(&d_noisy_image, sizeof(uchar) * h_channels * size);
    hipMemcpy(d_noisy_image, src_image, sizeof(uchar) * h_channels * size, hipMemcpyHostToDevice);

    // Only use the generic params for now
    GlobalConstants params;
    params.image_width = h_width;
    params.image_height = h_height;
    params.image_data = d_noisy_image;
    params.image_channels = h_channels;

    params.patch_size = h_fst_step_params.patch_size;
    params.searching_window_size = h_fst_step_params.searching_window_size;
    params.stripe = h_fst_step_params.stripe;
    params.max_group_size = h_fst_step_params.max_group_size;
    params.distance_threshold_1 = h_fst_step_params.distance_threshold_1;
    params.distance_threshold_2 = h_fst_step_params.distance_threshold_2;
    params.sigma = h_fst_step_params.sigma;
    params.lambda_3d = h_fst_step_params.lambda_3d;
    params.beta = h_fst_step_params.beta;
    printf("params: %d, %d\n", params.image_width, params.image_height);

    err = hipMemcpyToSymbol(HIP_SYMBOL(cu_const_params), &params, sizeof(GlobalConstants));

    printf("%s\n", hipGetErrorString(err));
}

/*
 * Initialize image stats and allocate memory
 */
void Bm3d::copy_image_to_device(uchar *src_image,
                                int width,
                                int height,
                                int channels) {
    // set width and height

}

void Bm3d::free_device_params() {
    if (d_noisy_image) {
        hipFree(d_noisy_image);
    }
}

/*
 * Take an image and run the algorithm to denoise.
 */
void Bm3d::denoise(uchar *src_image,
                   uchar *dst_image,
                   int width,
                   int height,
                   int channels,
                   int step,
                   int verbose = 1) {
    h_width = width;
    h_height = height;
    h_channels = channels;
    set_device_param(src_image);
    // first step
    test_cufft(src_image, dst_image);
    // arrange_block(src_image);
    // second step

    // copy image from device to host
    free_device_params();
}

/*
 * Perform the first step denoise
 */
void Bm3d::denoise_fst_step() {

}

/*
 * Perform the second step denoise
 */
void Bm3d::denoise_2nd_step() {

}

void Bm3d::run_kernel() {
    kernel<<<1,1>>>();
}

void Bm3d::test_cufft(uchar* src_image, uchar* dst_image) {
    Stopwatch init_time;
    Stopwatch exec_time;
    init_time.start();
    int size = h_width * h_height;
    int patch_size = 16;
    int group_size = 4;
    int batch = size / (patch_size*patch_size*group_size);

    hipfftHandle plan;
    hipfftHandle plan1D;
    uchar *h_data;
    uchar *d_data;
    hipMalloc(&d_data, sizeof(uchar) * size);

    hipMalloc(&h_data, sizeof(uchar) * size);
    hipMemcpy(h_data, src_image, sizeof(uchar) * size, hipMemcpyHostToDevice);

    hipfftComplex *data;
    hipMalloc(&data, sizeof(hipfftComplex) * size);
    int n[2] = {16,16};

    if(hipfftPlanMany(&plan, 2, n,
                     NULL, 1, 0,
                     NULL, 1, 0,
                     HIPFFT_C2C, size/256) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT Plan error: Plan failed");
        return;
    }
    if(hipfftPlan1d(&plan1D, patch_size*patch_size*group_size,
                     HIPFFT_C2C, batch) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT Plan error: Plan failed");
        return;
    }
    init_time.stop();
    exec_time.start();
    // get input in shape
    dim3 dimBlock(16,16);
    dim3 dimGrid(h_width/16, h_height/16);
    real2complex<<<dimGrid, dimBlock>>>(h_data, data);

    if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }

    if (hipfftExecC2C(plan1D, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }

    if (hipfftExecC2C(plan1D, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }

    if (hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return;
    }
    complex2real<<<dimGrid, dimBlock>>>(data, d_data, n[0]*n[1]);
    hipMemcpy(dst_image, d_data, size * sizeof(uchar), hipMemcpyDeviceToHost);
    if (hipGetLastError() != hipSuccess) {
        fprintf(stderr, "Cuda error: Failed results copy\n");
        return;
    }
    exec_time.stop();
    printf("Init: %f\n", init_time.getSeconds());
    printf("Exec: %f\n", exec_time.getSeconds());
    for (int i=0;i<size;i++) {
        printf("%d: (%zu, %zu)\n", i, src_image[i], dst_image[i]);
    }
}

/*
 *  arrange_block - according to the stacked patch indices, fill in the transformed
 *                  data array for 2D DCT. Input is an array of uint2, every N uint2
 *                  is a group. This kernel will put each group into an continuous array
 *                  of hipfftComplex num with x component to be the value, y component to be 0.f
 */
void Bm3d::arrange_block(uchar* src_image) {
    // initialize stacked patch indices which is a uint2 indices, each entry is the top
    // left indices of the patch
    int size = 8;
    int group_size = 2;
    int patch_size = 4;
    uint2 *h_stacks;
    uint2 *d_stacks;
    hipfftComplex *data_stack;

    h_stacks = (uint2*)malloc(sizeof(uint2) * size);
    for (int i=0;i<size;i++) {
        h_stacks[i].x = i*size;
        h_stacks[i].y = 0;
        for (int j=0;j<patch_size;j++) {
            for (int k=0;k<patch_size;k++) {
                printf("Image id: %d, %d\n", j*h_width + i*patch_size + k, src_image[idx2(i*size + k, j, h_width)]);
            }
        }
    }
    hipMalloc(&d_stacks, sizeof(uint2) * size);
    hipMemcpy(d_stacks, h_stacks, sizeof(uint2) * size, hipMemcpyHostToDevice);

    hipMalloc(&data_stack, sizeof(hipfftComplex) * size * patch_size * patch_size);

    // group per block, each pixel maps to one thread
    dim3 dimBlock(patch_size, patch_size);
    dim3 dimGrid(size/group_size);
    fill_data<<<dimGrid, dimBlock>>>(d_stacks, data_stack, size, patch_size, group_size);

}
